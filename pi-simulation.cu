#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <random>
using namespace std;


 //Fonction d'équation du cercle avec __global__ pour utiliser sur host ou device
__global__ void
cercle(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i]*A[i] + B[i]*B[i];
    }
}

int main(void) {
	
	int numElements;
	cout << "---Estimation de pi par méthode de monte carlo sur GPU---" << endl;
    cout << "Entrez le nombre de simulations que vous voulez faire : " << endl;
    cin >> numElements;
    
    
    size_t size = numElements * sizeof(float);

	//allocation de la taille des vecteurs dans le host (CPU)
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // simulation
    cout << "simulation..." << endl;
    mt19937 G;
    uniform_real_distribution<float> U(-1.,1.)
    for (int i = 0; i < numElements; ++i)
		{
        h_A[i] = U(G);
        h_B[i] = U(G);
		}

    // allocation de la taille des vecteurs dans le device (GPU)
    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);
    float *d_B = NULL;
    hipMalloc((void **)&d_B, size);
    float *d_C = NULL;
    hipMalloc((void **)&d_C, size);


    // Copie host -> device (input)
    cout << "Copie host -> device (input)" << endl;
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
   cercle<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Copie device -> host (output)
    cout << "Copie device -> host (output)" << endl;
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Vérification
    float est_pi =0.;
    for (int i = 0; i < numElements; ++i) est_pi + = h_C[i]
	cout << "estimation de pi : " << est_pi*4;


    // Free device global memory
    hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    printf("Done\n");
    return 0;
}
